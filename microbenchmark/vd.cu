
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define N (1024*10)
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(float *a, float *b, float *c)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
	
    // Make sure we do not go out of bounds
    if (id < N)
        c[id] = a[id] + b[id];
}

int main( int argc, char* argv[] )
{
    // Size of vectors
    //int n = 10000;
	
    // Host input vectors
    float *h_a;
    float *h_b;
    //Host output vector
    float *h_c;
	
    // Device input vectors
    float *d_a;
    float *d_b;
    //Device output vector
    float *d_c;
	
    // Size, in bytes, of each vector
    size_t bytes = N*sizeof(float);
	
    // Allocate memory for each vector on host
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
	// Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
	
    int i;
    // Initialize vectors on host
    for( i = 0; i < N; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }
	
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
	
    int blockSize, gridSize;
	
    // Number of threads in each thread block
    blockSize = 1024;
	
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)N/blockSize);
	
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c);
	
    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
	
    // Sum up vector c and print result divided by n, this should equal 1 within error
    float sum = 0;
    for(i=0; i< N; i++) {
        sum += h_c[i];
		//printf("h_c[%d]=%f\n",i,h_c[i]);
	}
	//printf("Sum is %f\n",sum);
    printf("final result: %f\n", sum/N);
	
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
	
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
	return 0;
}
