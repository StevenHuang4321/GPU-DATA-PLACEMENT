#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define N (1024)

texture<int> tex_a;
texture<int> tex_b;
texture<int> tex_c;

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(int *c)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
   // if (id < N) {
		
		c[id] = tex1Dfetch(tex_a,id) + tex1Dfetch(tex_b,id);
//	}
        
}

int main( int argc, char* argv[] )
{
    // Size of vectors
    //int n = 10000;
	
    // Host input vectors
    int *h_a;
    int *h_b;
    //Host output vector
    int *h_c;
	
    // Device input vectors
    int *d_a;
    int *d_b;
    //Device output vector
    int *d_c;
	
    // Size, in bytes, of each vector
    size_t bytes = N*sizeof(int);
	
    // Allocate memory for each vector on host
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);
	// Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
	
    int i;
    // Initialize vectors on host
    for( i = 0; i < N; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
		//h_c[i] = 0.0f;
    }
	// bind to texture memory
	hipBindTexture( NULL, tex_a,
					 d_a,
					 bytes );
	hipBindTexture( NULL, tex_b,
					 d_b,
					 bytes );
	hipBindTexture( NULL, tex_c,
					 d_c,
					 bytes );
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
	
	
    int blockSize, gridSize;
	
    // Number of threads in each thread block
    blockSize = 1024;
	
    // Number of thread blocks in grid
    gridSize = (int)ceil((int)N/blockSize);
	
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_c);
	
    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
	
    // Sum up vector c and print result divided by n, this should equal 1 within error
    int sum = 0;
    for(i=0; i<N; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/N);
	
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
	
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
	return 0;
}

