#include "hip/hip_runtime.h"
#define N 1024
__global__ void VectorAdd(float* A, float* B, float* C)
{
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}

__global__ void MatAdd(float A[N][N],float* B[N][N],float*C[N][N])
{
	int i = blockIdx.x * blcokDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i<N && j<N)
		C[i][j] = A[i][j] + B[i][j];
}

int main()
{
	VectorAdd<<<1,N,>>>(A,B,C);

	dim3 numBlocks( N/threadsPerBlock.x, N/threadsPerBlock.y);
	dim3 threadsPerBlock(16,16);

	MatAdd<<< numBlocks, threadsPerBlock >>>(A,B,C);
	__syncthreads();
	return 0;
}