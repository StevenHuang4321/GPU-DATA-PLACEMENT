#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
* This sample implements a separable convolution filter
* of a 2D image with an arbitrary kernel.
*/

// CUDA runtime
#include <hip/hip_runtime.h>

// Utilities and system includes
//#include <helper_functions.h>
#include <hip/hip_runtime.h>
#include "../include/common.h"
//#include <ctime.h>
#include <time.h>

#define KERNEL_RADIUS 8
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

__constant__ float c_Kernel[KERNEL_LENGTH];

void setConvolutionKernel(float *h_Kernel)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}


////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
#define   ROWS_BLOCKDIM_X 16
#define   ROWS_BLOCKDIM_Y 4
#define ROWS_RESULT_STEPS 8
#define   ROWS_HALO_STEPS 1

__global__ void convolutionRowsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    //Offset to the left halo edge
    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Load main data
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
    }

    //Load left halo
#pragma unroll

    for (int i = 0; i < ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Load right halo
#pragma unroll

    for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        float sum = 0;

#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
        }

        d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}

void convolutionRowsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH
)
{
    assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
    assert(imageH % ROWS_BLOCKDIM_Y == 0);

    dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), imageH / ROWS_BLOCKDIM_Y);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

    convolutionRowsKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageW
    );
    getLastCudaError("convolutionRowsKernel() execution failed\n");
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

    float
    *h_Kernel,
    *h_Input,
    *h_Buffer,
	*h_Output;

    float
    *d_Input,
    *d_Buffer;

    const int imageW = 3072;
    const int imageH = 3072;
    h_Kernel    = (float *)malloc(KERNEL_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_Output = (float *)malloc(imageW * imageH * sizeof(float));
    srand(200);

    for (unsigned int i = 0; i < KERNEL_LENGTH; i++)
    {
        h_Kernel[i] = (float)(rand() % 16);
    }

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
        h_Input[i] = (float)(rand() % 16);
    }


    hipMalloc((void **)&d_Input,   imageW * imageH * sizeof(float));
    hipMalloc((void **)&d_Buffer , imageW * imageH * sizeof(float));
    setConvolutionKernel(h_Kernel);
	hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice);

    convolutionRowsGPU(
            d_Buffer,
            d_Input,
            imageW,
            imageH
    );


	hipMemcpy(d_Buffer, h_Output, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_Buffer);
    hipFree(d_Input);
    free(h_Buffer);
    free(h_Input);
    free(h_Kernel);
	free(h_Output);
  
    hipDeviceReset();
    exit(EXIT_SUCCESS);
}
