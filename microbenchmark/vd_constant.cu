#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define N 1024

__constant__ double c_a[N];
__constant__ double c_b[N];

void setConstant(double *h_a, double *h_b)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_a), h_a, N * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_b), h_b, N * sizeof(double));
}
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *c)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
	
    // Make sure we do not go out of bounds
    if (id < N)
        c[id] = c_a[id] + c_b[id];
}

int main( int argc, char* argv[] )
{
    // Size of vectors
    //int n = 10000;
	
    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;
	
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
	
    // Size, in bytes, of each vector
    size_t bytes = N*sizeof(double);
	
    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
	// Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
	
    int i;
    // Initialize vectors on host
    for( i = 0; i < N; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }
	
    // Copy host vectors to device
    checkCudaErrors(hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice));
	
    int blockSize, gridSize;
	
    // Number of threads in each thread block
    blockSize = 1024;
	
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)N/blockSize);
	
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_c);
	
    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
	
    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    for(i=0; i<N; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/N);
	
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
	
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
	return 0;
}
