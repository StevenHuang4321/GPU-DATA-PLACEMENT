#include "hip/hip_runtime.h"

#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <list>
#include <map>
#include <math.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <algorithm>
#include <iterator>
#include <fstream>
#include "../include/common.h"
#define K 1
using namespace std;


#define spmv_NBLOCKS 12*8*21 //22
#define spmv_BLOCK_SIZE 256
#define WARP_SIZE 32
texture<float,1,hipReadModeElementType> tex_vec;
texture<int,1,hipReadModeElementType> tex_cols;
texture<float,1,hipReadModeElementType> tex_val;
static const double MAX_RELATIVE_ERROR = .02;

static const int PAD_FACTOR = 16;


void fill(float *A, const int n, const float maxi)
{
  for (int j = 0; j < n; j++) 
  {
    A[j] = ((float) maxi * (rand() / (RAND_MAX + 1.0f)));
  }
}

void initRandomMatrix(int *cols, int *rowDelimiters, const int n, const int dim)
{
  int nnzAssigned = 0;

  // Figure out the probability that a nonzero should be assigned to a given
  // spot in the matrix
  double prob = (double)n / ((double)dim * (double)dim);

  // Seed random number generator
  srand48(2013);

  // Randomly decide whether entry i,j gets a value, but ensure n values
  // are assigned
  bool fillRemaining = false;
  for (int i = 0; i < dim; i++)
  {
    rowDelimiters[i] = nnzAssigned;
    for (int j = 0; j < dim; j++)
    {
      int numEntriesLeft = (dim * dim) - ((i * dim) + j);
      int needToAssign   = n - nnzAssigned;
      if (numEntriesLeft <= needToAssign) {
        fillRemaining = true;
      }
      if ((nnzAssigned < n && drand48() <= prob) || fillRemaining)
      {
        // Assign (i,j) a value
        cols[nnzAssigned] = j;
        nnzAssigned++;
      }
    }
  }
  // Observe the convention to put the number of non zeroes at the end of the
  // row delimiters array
  rowDelimiters[dim] = n;
  assert(nnzAssigned == n);
}

void convertToPadded(float *A, int *cols, int dim, int *rowDelimiters, 
                     float **newA_ptr, int **newcols_ptr, int *newIndices, 
                     int *newSize) 
{
  // determine total padded size and new row indices
  int paddedSize = 0;  
  int rowSize; 

  for (int i=0; i<dim; i++) 
  {    
    newIndices[i] = paddedSize; 
    rowSize = rowDelimiters[i+1] - rowDelimiters[i]; 
    if (rowSize % PAD_FACTOR != 0) 
    {
      rowSize += PAD_FACTOR - rowSize % PAD_FACTOR; 
    } 
    paddedSize += rowSize; 
  }
  *newSize = paddedSize; 
  newIndices[dim] = paddedSize; 

  hipHostMalloc(newA_ptr, paddedSize * sizeof(float)); 
  hipHostMalloc(newcols_ptr, paddedSize * sizeof(int)); 

  float *newA = *newA_ptr; 
  int *newcols = *newcols_ptr; 

  memset(newA, 0, paddedSize * sizeof(float)); 

  // fill newA and newcols
  for (int i=0; i<dim; i++) 
  {
    for (int j=rowDelimiters[i], k=newIndices[i]; j<rowDelimiters[i+1]; 
        j++, k++) 
    {
      newA[k] = A[j]; 
      newcols[k] = cols[j]; 
    }
  }
}

void spmvCpu(const float *val, const int *cols, const int *rowDelimiters, 
	     const float *vec, int dim, float *out) 
{
  for (int i=0; i<dim; i++) 
  {
    float t = 0; 
    for (int j = rowDelimiters[i]; j < rowDelimiters[i + 1]; j++)
    {
      int col = cols[j]; 
      t += val[j] * vec[col];//tex1Dfetch(tex_vec,col);
    }    
    out[i] = t; 
  }
}

void spmv_verifyResults(const float *cpuResults, const float *gpuResults,
                   const int size) 
{
  bool passed = true; 
  for (int i = 0; i < size; i++)
  {
    if (fabs(cpuResults[i] - gpuResults[i]) / cpuResults[i] 
        > MAX_RELATIVE_ERROR) 
    {
      cout << "Failed! Mismatch at i: "<< i << " ref: " << cpuResults[i] <<
          " dev: " << gpuResults[i] << endl;
      return;
    }
  }

  cout << "spmv passed" << endl;
}
__global__ void 
spmv_kernel(volatile float*  val,
                       const int* __restrict__ cols,
                       const int    * rowDelimiters,
                       volatile float  * vec,
                       const int dim, float * out)
{
  // Thread ID in block
  int t = threadIdx.x; 
  // Thread ID within warp
  int id = t & (WARP_SIZE-1);
  int warpsPerBlock = blockDim.x / WARP_SIZE;
  // One row per warp
  int myRow = (blockIdx.x * warpsPerBlock) + (t / WARP_SIZE);
//  __shared__ int rowDeli[spmv_BLOCK_SIZE/WARP_SIZE+1];
  __shared__ volatile float partialSums[spmv_BLOCK_SIZE];
/*  if (threadIdx.x<spmv_BLOCK_SIZE/WARP_SIZE+1)
 rowDeli[threadIdx.x]=rowDelimiters[myRow+threadIdx.x];

__syncthreads();
*/
  if (myRow < dim) 
  {
    int warpStart = rowDelimiters[myRow];
    int warpEnd = rowDelimiters[myRow+1];
    float mySum = 0;
    for (int j = warpStart + id; j < warpEnd; j += WARP_SIZE)
    {
      int col = tex1Dfetch(tex_cols,j); 
      mySum += val[j]*vec[col];//tex1Dfetch(tex_val,j) *vec[col];
    }
    partialSums[t] = mySum;

    // Reduce partial sums
    if (id < 16) partialSums[t] += partialSums[t+16];
    if (id <  8) partialSums[t] += partialSums[t+ 8];
    if (id <  4) partialSums[t] += partialSums[t+ 4];
    if (id <  2) partialSums[t] += partialSums[t+ 2];
    if (id <  1) partialSums[t] += partialSums[t+ 1];

    // Write result 
    if (id == 0)
    {
      out[myRow] = partialSums[t];
    }
  }
}

int main(int argc, char **argv) {
  hipSetDevice(2);
  srand(2013);
  float *h_spmv_val, *h_spmv_valPad;
  int *h_spmv_cols, *h_spmv_colsPad;
  int *h_rowDelimiters, *h_rowDelimitersPad;
  float *h_spmv_vec, *h_spmv_out, *spmv_refOut;
  int spmv_nItems, nItemsPadded, spmv_numRows;

  spmv_numRows = spmv_NBLOCKS * (spmv_BLOCK_SIZE/WARP_SIZE);
  spmv_nItems = spmv_numRows * spmv_numRows / 10; // 1% of entries will be non-zero
  float maxval = 200.0;
  hipHostMalloc(&h_spmv_val, spmv_nItems * sizeof(float)); 
  hipHostMalloc(&h_spmv_cols, spmv_nItems * sizeof(int)); 
  hipHostMalloc(&h_rowDelimiters, (spmv_numRows + 1) * sizeof(int)); 
  fill(h_spmv_val, spmv_nItems, maxval); 
  initRandomMatrix(h_spmv_cols, h_rowDelimiters, spmv_nItems, spmv_numRows);

  // Set up remaining host data
  int paddedSize = spmv_numRows + (PAD_FACTOR - spmv_numRows % PAD_FACTOR);
  hipHostMalloc(&h_spmv_vec, spmv_numRows * sizeof(float)) ;
  spmv_refOut = new float[spmv_numRows];
  hipHostMalloc(&h_rowDelimitersPad, (spmv_numRows + 1) * sizeof(int)); 
  fill(h_spmv_vec, spmv_numRows, maxval);

  hipHostMalloc(&h_spmv_out, paddedSize * sizeof(float)); 
  convertToPadded(h_spmv_val, h_spmv_cols, spmv_numRows, h_rowDelimiters, &h_spmv_valPad,
          &h_spmv_colsPad, h_rowDelimitersPad, &nItemsPadded);

  // Compute reference solution
  spmvCpu(h_spmv_val, h_spmv_cols, h_rowDelimiters, h_spmv_vec, spmv_numRows, spmv_refOut);

  float *d_spmv_val, *d_spmv_vec, *d_spmv_out;
  int *d_spmv_cols, *d_rowDelimiters;

  // Allocate device memory
  hipMalloc(&d_spmv_val,  spmv_nItems * sizeof(float));
  hipMalloc(&d_spmv_cols, spmv_nItems * sizeof(int));
  hipMalloc(&d_spmv_vec,  spmv_numRows * sizeof(float));
  hipMalloc(&d_spmv_out,  spmv_numRows * sizeof(float));
  hipMalloc(&d_rowDelimiters, (spmv_numRows+1) * sizeof(int));

  // Transfer data to device
  hipMemcpy(d_spmv_val, h_spmv_val,   spmv_nItems * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_spmv_cols, h_spmv_cols, spmv_nItems * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_spmv_vec, h_spmv_vec, spmv_numRows * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_rowDelimiters, h_rowDelimiters, (spmv_numRows+1) * sizeof(int), hipMemcpyHostToDevice);
 
  hipBindTexture(0,tex_vec,d_spmv_vec,spmv_numRows * sizeof(float));
  hipBindTexture(0,tex_cols,d_spmv_cols,spmv_nItems * sizeof(int));
  hipBindTexture(0,tex_val,d_spmv_val,spmv_nItems * sizeof(float)); 
  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float kernel_time = 0.0f;

  hipEventRecord(kernel_start, 0);

  // Setup thread configuration
  int spmv_grid = (int) ceil(spmv_numRows / (float)(spmv_BLOCK_SIZE / WARP_SIZE));

  spmv_kernel <<<spmv_grid, spmv_BLOCK_SIZE>>>
  (d_spmv_val, d_spmv_cols, d_rowDelimiters, d_spmv_vec, spmv_numRows, d_spmv_out);

  hipDeviceSynchronize();

  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);

  // get elapsed time
  kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
  kernel_time *= 1.e-3; // Convert to seconds
  
  cout << "kernel exe time: " << kernel_time << endl;
  hipMemcpy(h_spmv_out, d_spmv_out, spmv_numRows * sizeof(float), hipMemcpyDeviceToHost);
  spmv_verifyResults(spmv_refOut, h_spmv_out, spmv_numRows);

  return 0;
}

