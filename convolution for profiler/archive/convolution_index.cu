#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
* This sample implements a separable convolution filter
* of a 2D image with an arbitrary kernel.
*/

// CUDA runtime
#include <hip/hip_runtime.h>
#include <stdio.h>
// Utilities and system includes
//#include <helper_functions.h>
#include <hip/hip_runtime.h>
#include "../include/common.h"
//#include <ctime.h>
#include <time.h>

#define KERNEL_RADIUS 8
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

//__constant__ float c_Kernel[KERNEL_LENGTH];

/*void setConvolutionKernel(float *h_Kernel)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}*/


////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
#define   ROWS_BLOCKDIM_X 16
#define   ROWS_BLOCKDIM_Y 4
#define ROWS_RESULT_STEPS 8
#define   ROWS_HALO_STEPS 1

__global__ void convolutionRowsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch,
    float *c_Kernel,
    int *c_Kernel_index,
    int *d_Src_index
)
{
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    //Offset to the left halo edge
    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

    d_Src += baseY * pitch + baseX;
   // d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Load main data
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
        if(blockIdx.x==1&&blockIdx.y==1&&threadIdx.x<4&&threadIdx.y<4)
        atomicAdd(&d_Src_index[(threadIdx.x+threadIdx.y*4)*imageW*imageH+baseY*pitch+baseX+i*ROWS_BLOCKDIM_X],1);//+=1;
    }

    //Load left halo
#pragma unroll

    for (int i = 0; i < ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
        if(blockIdx.x==1&&blockIdx.y==1&&threadIdx.x<4&&threadIdx.y<4){int index=threadIdx.x+threadIdx.y*4;
        atomicAdd(&d_Src_index[index*imageW*imageH+baseY*pitch+baseX+i*ROWS_BLOCKDIM_X],1);}
 }

    //Load right halo
#pragma unroll

    for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
        if(blockIdx.x==1&&blockIdx.y==1&&threadIdx.x<4&&threadIdx.y<4)
        atomicAdd(&d_Src_index[(threadIdx.x+threadIdx.y*4)*imageW*imageH+baseY*pitch+baseX+i*ROWS_BLOCKDIM_X],1);  
  }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        float sum = 0;

#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
       if(blockIdx.x==0&&blockIdx.y==0&&threadIdx.x<8&&threadIdx.y<8)
        c_Kernel_index[(threadIdx.y*8+threadIdx.x)*KERNEL_LENGTH+KERNEL_RADIUS-j]+=1;
        //atomicAdd(&c_Kernel_index[(threadIdx.y*8+threadIdx.x)*KERNEL_LENGTH+KERNEL_RADIUS-j],1);    

        }

        d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}

void convolutionRowsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    float *c_Kernel,
    int *c_Kernel_index,
    int *d_Src_index
)
{
    assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
    assert(imageH % ROWS_BLOCKDIM_Y == 0);

    dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), imageH / ROWS_BLOCKDIM_Y);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

    convolutionRowsKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageW,
        c_Kernel,
        c_Kernel_index,
        d_Src_index
    );
    getLastCudaError("convolutionRowsKernel() execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
#define   COLUMNS_BLOCKDIM_X 16
#define   COLUMNS_BLOCKDIM_Y 8
#define COLUMNS_RESULT_STEPS 8
#define   COLUMNS_HALO_STEPS 1

__global__ void convolutionColumnsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch,
    float *c_Kernel
)
{
    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Main data
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * pitch];
    }

    //Upper halo
#pragma unroll

    for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Lower halo
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        float sum = 0;
#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
        }

        d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
    }
}

void convolutionColumnsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    float *c_Kernel
)
{
    assert(COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % COLUMNS_BLOCKDIM_X == 0);
    assert(imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0);

    dim3 blocks(imageW / COLUMNS_BLOCKDIM_X, imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);

    convolutionColumnsKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageW,
        c_Kernel
    );
    getLastCudaError("convolutionColumnsKernel() execution failed\n");
}

void convolutionRowCPU(
    float *h_Dst,
    float *h_Src,
    float *h_Kernel,
    int imageW,
    int imageH,
    int kernelR
)
{
    for (int y = 0; y < imageH; y++)
        for (int x = 0; x < imageW; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = x + k;

                if (d >= 0 && d < imageW)
                    sum += h_Src[y * imageW + d] * h_Kernel[kernelR - k];
            }

            h_Dst[y * imageW + x] = sum;
        }
}



////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(
    float *h_Dst,
    float *h_Src,
    float *h_Kernel,
    int imageW,
    int imageH,
    int kernelR
)
{
    for (int y = 0; y < imageH; y++)
        for (int x = 0; x < imageW; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = y + k;

                if (d >= 0 && d < imageH)
                    sum += h_Src[d * imageW + x] * h_Kernel[kernelR - k];
            }

            h_Dst[y * imageW + x] = sum;
        }
}



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // start logs
    printf("[%s] - Starting...\n", argv[0]);

    float
    *h_Kernel,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;

    float
    *d_Input,
    *d_Output,
    *d_Buffer,
    *c_Kernel;


    const int imageW = 3072;
    const int imageH = 3072;
    const int iterations = 16;

    struct timespec t1,t2;
    

    //Use command-line specified CUDA device, otherwise use device with highest Gflops/s
    //findCudaDevice(argc, (const char **)argv);

    

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    h_Kernel    = (float *)malloc(KERNEL_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));
    srand(200);

    for (unsigned int i = 0; i < KERNEL_LENGTH; i++)
    {
        h_Kernel[i] = (float)(rand() % 16);
    }

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
        h_Input[i] = (float)(rand() % 16);
    }

    printf("Allocating and initializing CUDA arrays...\n");
    checkCudaErrors(hipMalloc((void **)&d_Input,   imageW * imageH * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Output,  imageW * imageH * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Buffer , imageW * imageH * sizeof(float)));
    hipMalloc((void **)&c_Kernel, KERNEL_LENGTH*sizeof(float));     
    int *c_Kernel_index,*d_Src_index;
    checkCudaErrors(hipMalloc((void **)&d_Src_index,16*imageW*imageH*sizeof(int)));

    hipMalloc((void **)&c_Kernel_index,64*KERNEL_LENGTH*sizeof(int));
     hipMemcpy(c_Kernel, h_Kernel, KERNEL_LENGTH * sizeof(float),hipMemcpyHostToDevice);
       // setConvolutionKernel(h_Kernel);
    checkCudaErrors(hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice));

    printf("Running GPU convolution (%u identical iterations)...\n\n", iterations);

    for (int i = -1; i < iterations; i++)
    {
        //i == -1 -- warmup iteration
        if (i == 0)
        {
            checkCudaErrors(hipDeviceSynchronize());
            clock_gettime(CLOCK_MONOTONIC,&t1);
        }
        hipMemset(c_Kernel_index,0,64*KERNEL_LENGTH*sizeof(int));
    checkCudaErrors(hipMemset(d_Src_index,0,16*imageW*imageH*sizeof(int)));

        convolutionRowsGPU(
            d_Buffer,
            d_Input,
            imageW,
            imageH,
            c_Kernel,
            c_Kernel_index,
            d_Src_index
        );

        convolutionColumnsGPU(
            d_Output,
            d_Buffer,
            imageW,
            imageH,
            c_Kernel
        );
   //printf("%d\n",i);
   checkCudaErrors(hipDeviceSynchronize()); 
   }

    checkCudaErrors(hipDeviceSynchronize());
    clock_gettime(CLOCK_MONOTONIC,&t2);
    double gpuTime = ((t2.tv_sec-t1.tv_sec)+ (t2.tv_nsec-t1.tv_nsec)/1.e9)/ (double)iterations;
    printf("convolutionSeparable, Throughput = %.4f MPixels/sec, Time = %.5f s, Size = %u Pixels, NumDevsUsed = %i, Workgroup = %u\n",
           (1.0e-6 * (double)(imageW * imageH)/ gpuTime), gpuTime, (imageW * imageH), 1, 0);

    printf("\nReading back GPU results...\n\n");
    checkCudaErrors(hipMemcpy(h_OutputGPU, d_Output, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost));
    int *h_index=(int *)malloc(KERNEL_LENGTH*16*sizeof(int));
    int *h_Src_index=(int *)malloc(16*imageW*imageH*sizeof(int));
     hipMemcpy(h_Src_index,d_Src_index,16*imageW*imageH*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(h_index,c_Kernel_index,KERNEL_LENGTH*16*sizeof(int),hipMemcpyDeviceToHost);
   FILE *f1=fopen("c_Kernel_D2.txt","w");
   for(int ii=0;ii<16;ii++){
     for(int jj=0;jj<KERNEL_LENGTH;jj++)
    {if(h_index[ii*KERNEL_LENGTH+jj]!=0)
      fprintf(f1,"%d,%d ;",jj,h_index[ii*KERNEL_LENGTH+jj]);
   }
  fprintf(f1,"\n"); 
}
  FILE *f2=fopen("d_Src_D2.txt","w");
 for(int ii=0;ii<16;ii++){
    // fprintf(f2,"\n");
     for(int jj=0;jj<imageW*imageH;jj++)
    {if(h_Src_index[ii*imageW*imageH+jj]!=0)
     fprintf(f2,"%d,%d ;",jj,h_Src_index[ii*imageW*imageH+jj]);
   }
fprintf(f2,"\n");
}

   printf("Checking the results...\n");
    printf(" ...running convolutionRowCPU()\n");
    convolutionRowCPU(
        h_Buffer,
        h_Input,
        h_Kernel,
        imageW,
        imageH,
        KERNEL_RADIUS
    );

    printf(" ...running convolutionColumnCPU()\n");
    convolutionColumnCPU(
        h_OutputCPU,
        h_Buffer,
        h_Kernel,
        imageW,
        imageH,
        KERNEL_RADIUS
    );

    printf(" ...comparing the results\n");
    double sum = 0, delta = 0;

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
        delta += (h_OutputGPU[i] - h_OutputCPU[i]) * (h_OutputGPU[i] - h_OutputCPU[i]);
        sum   += h_OutputCPU[i] * h_OutputCPU[i];
    }

    double L2norm = sqrt(delta / sum);
    printf(" ...Relative L2 norm: %E\n\n", L2norm);
    printf("Shutting down...\n");


    checkCudaErrors(hipFree(d_Buffer));
    checkCudaErrors(hipFree(d_Output));
    checkCudaErrors(hipFree(d_Input));
    free(h_OutputGPU);
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Kernel);

    

    hipDeviceReset();

    if (L2norm > 1e-6)
    {
        printf("Test failed!\n");
        exit(EXIT_FAILURE);
    }

    printf("Test passed\n");
    printf("Profiling results saved to \"d_Src_D2.txt\" and \"c_Kernel_D2.txt\"\n");
    printf("Please use analysis.py to analysis them\n");
    exit(EXIT_SUCCESS);
}
