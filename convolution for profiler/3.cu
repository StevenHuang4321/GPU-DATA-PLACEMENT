#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
* This sample implements a separable convolution filter
* of a 2D image with an arbitrary kernel.
*/

// CUDA runtime
#include <hip/hip_runtime.h>

// Utilities and system includes
//#include <helper_functions.h>
#include <hip/hip_runtime.h>
#include "../include/common.h"
//#include <ctime.h>
#include <time.h>

#define KERNEL_RADIUS 8 
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

__constant__ float c_Kernel[KERNEL_LENGTH];
texture<float,2,hipReadModeElementType> texRef;

void setConvolutionKernel(float *h_Kernel)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}


////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
#define   ROWS_BLOCKDIM_X 16
#define   ROWS_BLOCKDIM_Y 4
#define ROWS_RESULT_STEPS 8
#define   ROWS_HALO_STEPS 1

__global__ void convolutionRowsKernel(
    float *d_Dst,
    //float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    //Offset to the left halo edge
    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

    //d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Load main data
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = tex2D(texRef,baseX+i*ROWS_BLOCKDIM_X,baseY);//d_Src[i * ROWS_BLOCKDIM_X];
    }

    //Load left halo
#pragma unroll

    for (int i = 0; i < ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? tex2D(texRef,baseX+i*ROWS_BLOCKDIM_X,baseY):0;//d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Load right halo
//#pragma unroll
 //   printf("%d,%d\n",ROWS_HALO_STEPS,ROWS_HALO_STEPS);
    for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? tex2D(texRef,baseX+i*ROWS_BLOCKDIM_X,baseY):0;//d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        float sum = 0;

#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
        }

        d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}

void convolutionRowsGPU(
    float *d_Dst,
 //   float *d_Src,
    int imageW,
    int imageH
)
{
    assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
    assert(imageH % ROWS_BLOCKDIM_Y == 0);

    dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), imageH / ROWS_BLOCKDIM_Y);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

    convolutionRowsKernel<<<blocks, threads>>>(
        d_Dst,
       // d_Src,
        imageW,
        imageH,
        imageW
    );
    getLastCudaError("convolutionRowsKernel() execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
#define   COLUMNS_BLOCKDIM_X 16
#define   COLUMNS_BLOCKDIM_Y 8
#define COLUMNS_RESULT_STEPS 8
#define   COLUMNS_HALO_STEPS 1

__global__ void convolutionColumnsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{
    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Main data
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * pitch];
    }

    //Upper halo
#pragma unroll

    for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Lower halo
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        float sum = 0;
#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
        }

        d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
    }
}

void convolutionColumnsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH
)
{
    assert(COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % COLUMNS_BLOCKDIM_X == 0);
    assert(imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0);

    dim3 blocks(imageW / COLUMNS_BLOCKDIM_X, imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);

    convolutionColumnsKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageW
    );
    getLastCudaError("convolutionColumnsKernel() execution failed\n");
}

void convolutionRowCPU(
    float *h_Dst,
    float *h_Src,
    float *h_Kernel,
    int imageW,
    int imageH,
    int kernelR
)
{
    for (int y = 0; y < imageH; y++)
        for (int x = 0; x < imageW; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = x + k;

                if (d >= 0 && d < imageW)
                    sum += h_Src[y * imageW + d] * h_Kernel[kernelR - k];
            }

            h_Dst[y * imageW + x] = sum;
        }
}



////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(
    float *h_Dst,
    float *h_Src,
    float *h_Kernel,
    int imageW,
    int imageH,
    int kernelR
)
{
    for (int y = 0; y < imageH; y++)
        for (int x = 0; x < imageW; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = y + k;

                if (d >= 0 && d < imageH)
                    sum += h_Src[d * imageW + x] * h_Kernel[kernelR - k];
            }

            h_Dst[y * imageW + x] = sum;
        }
}



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // start logs
//    printf("[%s] - Starting...\n", argv[0]);

    float
    *h_Kernel,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;

    float
    *d_Input,
    *d_Output,
    *d_Buffer;


    const int imageW = 3072;
    const int imageH = 3072;
    const int iterations = 16;

//    struct timespec t1,t2;
    

    //Use command-line specified CUDA device, otherwise use device with highest Gflops/s
    //findCudaDevice(argc, (const char **)argv);

    

//    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
//    printf("Allocating and initializing host arrays...\n");
    h_Kernel    = (float *)malloc(KERNEL_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));
    srand(200);

    for (unsigned int i = 0; i < KERNEL_LENGTH; i++)
    {
        h_Kernel[i] = (float)(rand() % 16);
    }

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
        h_Input[i] = (float)(rand() % 16);
    }

//    printf("Allocating and initializing CUDA arrays...\n");
//    checkCudaErrors(hipMalloc((void **)&d_Input,   imageW * imageH * sizeof(float)));
//    checkCudaErrors(hipMalloc((void **)&d_Output,  imageW * imageH * sizeof(float)));
//    checkCudaErrors(hipMalloc((void **)&d_Buffer , imageW * imageH * sizeof(float)));
	hipMalloc((void **)&d_Input,   imageW * imageH * sizeof(float));
	hipMalloc((void **)&d_Output,  imageW * imageH * sizeof(float));
	hipMalloc((void **)&d_Buffer , imageW * imageH * sizeof(float));
    
    setConvolutionKernel(h_Kernel);
    //checkCudaErrors(hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice));
    
     hipChannelFormatDesc channelDesc =  hipCreateChannelDesc<float>();
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, imageW, imageH);

    // Copy to device memory some data located at address h_data
    // in host memory 
    hipMemcpyToArray(cuArray, 0, 0, h_Input,  imageW * imageH * sizeof(float),
                      hipMemcpyHostToDevice);

    // Set texture reference parameters
    texRef.addressMode[0] = hipAddressModeWrap;
    texRef.addressMode[1] = hipAddressModeWrap;
    texRef.filterMode     = hipFilterModePoint;
    

    // Bind the array to the texture reference
    hipBindTextureToArray(texRef, cuArray, channelDesc);

   
//    printf("Running GPU convolution (%u identical iterations)...\n\n", iterations);

    for (int i = -1; i < iterations; i++)
    {
        //i == -1 -- warmup iteration
        if (i == 0)
        {
 //           checkCudaErrors(hipDeviceSynchronize());
			hipDeviceSynchronize();
//            clock_gettime(CLOCK_MONOTONIC,&t1);
        }
//if(i==1) clock_gettime(CLOCK_MONOTONIC,&t1);

        convolutionRowsGPU(
            d_Buffer,
            //d_Input,
            imageW,
            imageH
        );
// checkCudaErrors(hipDeviceSynchronize());
		hipDeviceSynchronize();
//if(i==1)    clock_gettime(CLOCK_MONOTONIC,&t2);
        convolutionColumnsGPU(
            d_Output,
            d_Buffer,
            imageW,
            imageH
        );
    }

    //checkCudaErrors(hipDeviceSynchronize());
	hipDeviceSynchronize()
  //  clock_gettime(CLOCK_MONOTONIC,&t2);
  //  double gpuTime = ((t2.tv_sec-t1.tv_sec)+ (t2.tv_nsec-t1.tv_nsec)/1.e9);/// (double)iterations;
    //printf("convolutionSeparable, Throughput = %.4f MPixels/sec, Time = %.5f s, Size = %u Pixels, NumDevsUsed = %i, Workgroup = %u\n",
      //     (1.0e-6 * (double)(imageW * imageH)/ gpuTime), gpuTime, (imageW * imageH), 1, 0);

    //printf("\nReading back GPU results...\n\n");
    //checkCudaErrors(hipMemcpy(h_OutputGPU, d_Output, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost));
	hipMemcpy(h_OutputGPU, d_Output, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost);
    /*
	printf("Checking the results...\n");
    printf(" ...running convolutionRowCPU()\n");
    convolutionRowCPU(
        h_Buffer,
        h_Input,
        h_Kernel,
        imageW,
        imageH,
        KERNEL_RADIUS
    );

    printf(" ...running convolutionColumnCPU()\n");
    convolutionColumnCPU(
        h_OutputCPU,
        h_Buffer,
        h_Kernel,
        imageW,
        imageH,
        KERNEL_RADIUS
    );

    printf(" ...comparing the results\n");
    double sum = 0, delta = 0;

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
        delta += (h_OutputGPU[i] - h_OutputCPU[i]) * (h_OutputGPU[i] - h_OutputCPU[i]);
        sum   += h_OutputCPU[i] * h_OutputCPU[i];
    }

    double L2norm = sqrt(delta / sum);
    printf(" ...Relative L2 norm: %E\n\n", L2norm);
    printf("Shutting down...\n");
*/

 //   checkCudaErrors(hipFree(d_Buffer));
  //  checkCudaErrors(hipFree(d_Output));
   // checkCudaErrors(hipFree(d_Input));
	
	hipFree(d_Buffer);
    hipFree(d_Output);
    hipFree(d_Input);
    free(h_OutputGPU);
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Kernel);

    

    hipDeviceReset();
/*
    if (L2norm > 1e-6)
    {
        printf("Test failed!\n");
        exit(EXIT_FAILURE);
    }

    printf("Test passed\n");
*/
    exit(EXIT_SUCCESS);
}
