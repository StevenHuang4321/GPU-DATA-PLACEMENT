#include "hip/hip_runtime.h"

#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <list>
#include <map>
#include <math.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <algorithm>
#include <iterator>
#include <fstream>
#include "../include/common.h"

using namespace std;
#define mm_STREAM 0 
#define pf_STREAM 1

#define mm_BLOCK_SIZE 16
//#define mm_SUPER_BLOCKS_PER_SM 4
//int mm_SUPER_BLOCKS_PER_SM = 4;

#define iSizeMultiple 4 //must be multipes of 15

#define WA (4 * mm_BLOCK_SIZE) // Matrix A width
#define HA (4 * mm_BLOCK_SIZE) // Matrix A height
//#define WB (mm_SUPER_BLOCKS_PER_SM * mm_BLOCK_SIZE) // Matrix B width
#define WB (6 * 10 * mm_BLOCK_SIZE) // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height

#define mm_GRID_X (WC*iSizeMultiple/mm_BLOCK_SIZE)
#define mm_GRID_Y (HC*iSizeMultiple/mm_BLOCK_SIZE)
#define mm_NBLOCKS (mm_GRID_X*mm_GRID_Y)

#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]

void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void
computeGold(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
  for (unsigned int i = 0; i < hA; ++i)
    for (unsigned int j = 0; j < wB; ++j) {
      double sum = 0;
      for (unsigned int k = 0; k < wA; ++k) {
        double a = A[i * wA + k];
        double b = B[k * wB + j];
        sum += a * b;
      }
      C[i * wB + j] = (float)sum;
    }
}
#define pf_NBLOCKS 254*60 //16*6*2
#define pf_BLOCK_SIZE 256
#define STR_SIZE 256
#define HALO 1 // halo width along one direction when advancing to the next iteration

#define BENCH_PRINT
#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

int pf_rows, pf_cols;
int* pf_data;
int** wall;
int* pf_result;
#define M_SEED 9
int pyramid_height;

//#define BENCH_PRINT

  void
init(int argc, char** argv)
{

  pf_data = new int[pf_rows*pf_cols];
  wall = new int*[pf_rows];
  for(int n=0; n<pf_rows; n++)
    wall[n]=pf_data+pf_cols*n;
  pf_result = new int[pf_cols];

  int seed = M_SEED;
  srand(seed);

  for (int i = 0; i < pf_rows; i++)
  {
    for (int j = 0; j < pf_cols; j++)
    {
      wall[i][j] = rand() % 10;
    }
  }
}

  void 
fatal(char *s)
{
  fprintf(stderr, "error: %s\n", s);

}


void check_kernel(
    int iteration, 
    int *gpuWall,
    int *gpuSrc,
    int *gpuResults,
    int cols, 
    int rows,
    int startStep,
    int border,
    int dimGrid, 
    int dimBlock)
{



  for(int bx=0;bx<dimGrid;bx++){
    int prev[pf_BLOCK_SIZE];
    int result[pf_BLOCK_SIZE];
    for (int tx=0;tx<dimBlock;tx++)
    {     
      int small_block_cols = pf_BLOCK_SIZE-iteration*HALO*2;

      // calculate the boundary for the block according to 
      // the boundary of its small block
      int blkX = small_block_cols*bx-border;
      int blkXmax = blkX+pf_BLOCK_SIZE-1;

      // calculate the global thread coordination
      int xidx = blkX+tx;

      // effective range within this block that falls within 
      // the valid range of the input data
      // used to rule out computation outside the boundary.
      int validXmin = (blkX < 0) ? -blkX : 0;
      int validXmax = (blkXmax > cols-1) ? pf_BLOCK_SIZE-1-(blkXmax-cols+1) : pf_BLOCK_SIZE-1;

      int W = tx-1;
      int E = tx+1;

      W = (W < validXmin) ? validXmin : W;
      E = (E > validXmax) ? validXmax : E;

      bool isValid = IN_RANGE(tx, validXmin, validXmax);

      if(IN_RANGE(xidx, 0, cols-1)){
        prev[tx] = gpuSrc[xidx];
      }
    }
    //__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012

    for (int tx=0;tx<dimBlock;tx++)
    {    
      int small_block_cols = pf_BLOCK_SIZE-iteration*HALO*2;

      // calculate the boundary for the block according to 
      // the boundary of its small block
      int blkX = small_block_cols*bx-border;
      int blkXmax = blkX+pf_BLOCK_SIZE-1;

      // calculate the global thread coordination
      int xidx = blkX+tx;

      // effective range within this block that falls within 
      // the valid range of the input data
      // used to rule out computation outside the boundary.
      int validXmin = (blkX < 0) ? -blkX : 0;
      int validXmax = (blkXmax > cols-1) ? pf_BLOCK_SIZE-1-(blkXmax-cols+1) : pf_BLOCK_SIZE-1;

      int W = tx-1;
      int E = tx+1;

      W = (W < validXmin) ? validXmin : W;
      E = (E > validXmax) ? validXmax : E;

      bool isValid = IN_RANGE(tx, validXmin, validXmax);
      bool computed;
      for (int i=0; i<iteration ; i++){ 
        computed = false;
        if( IN_RANGE(tx, i+1, pf_BLOCK_SIZE-i-2) &&  \
            isValid){
          computed = true;
          int left = prev[W];
          int up = prev[tx];
          int right = prev[E];
          int shortest = MIN(left, up);
          shortest = MIN(shortest, right);
          int index = cols*(startStep+i)+xidx;
          result[tx] = shortest + gpuWall[index];

        }
        //__syncthreads();
        if(i==iteration-1)
          break;
        if(computed)	 //Assign the computation range
          prev[tx]= result[tx];
        // __syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
      }

      // update the global memory
      // after the last iteration, only threads coordinated within the 
      // small block perform the calculation and switch on ``computed''
      if (computed){
        //gpuResults[xidx]=result[tx];	
        if(gpuResults[xidx]!=result[tx]){printf("failed!%d %d %d %d\n",tx,bx,result[tx],gpuResults[xidx]);return;}	
      }
    }
  }
  printf("GOOD Kernel passed!\n");
  return;
}
__global__ void
mm_kernel( float* C, float* A, float* B, int wA, int wB)
{
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * mm_BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = mm_BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = mm_BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = mm_BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin;
      a <= aEnd;
      a += aStep, b += bStep) {

    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[mm_BLOCK_SIZE][mm_BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[mm_BLOCK_SIZE][mm_BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    AS(ty, tx) = A[a + wA * ty + tx];
    BS(ty, tx) = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll
    for (int k = 0; k < mm_BLOCK_SIZE; ++k)
      Csub += AS(ty, k) * BS(k, tx);

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * mm_BLOCK_SIZE * by + mm_BLOCK_SIZE * bx;

  C[c + wB * ty + tx] = Csub;
}

__global__ void pf_kernel(
    int iteration, 
    int *gpuWall,
    int *gpuSrc,
    int *gpuResults,
    int cols, 
    int rows,
    int startStep,
    int border)
{

  __shared__ int prev[pf_BLOCK_SIZE];
  __shared__ int result[pf_BLOCK_SIZE];

  int bx = blockIdx.x;
  int tx=threadIdx.x;


  int small_block_cols = pf_BLOCK_SIZE-iteration*HALO*2;

  // calculate the boundary for the block according to 
  // the boundary of its small block
  int blkX = small_block_cols*bx-border;
  int blkXmax = blkX+pf_BLOCK_SIZE-1;

  // calculate the global thread coordination
  int xidx = blkX+tx;

  // effective range within this block that falls within 
  // the valid range of the input data
  // used to rule out computation outside the boundary.
  int validXmin = (blkX < 0) ? -blkX : 0;
  int validXmax = (blkXmax > cols-1) ? pf_BLOCK_SIZE-1-(blkXmax-cols+1) : pf_BLOCK_SIZE-1;

  int W = tx-1;
  int E = tx+1;

  W = (W < validXmin) ? validXmin : W;
  E = (E > validXmax) ? validXmax : E;

  bool isValid = IN_RANGE(tx, validXmin, validXmax);

  if(IN_RANGE(xidx, 0, cols-1)){
    prev[tx] = gpuSrc[xidx];
  }
  __syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
  bool computed;
  for (int i=0; i<iteration ; i++){ 
    computed = false;
    if( IN_RANGE(tx, i+1, pf_BLOCK_SIZE-i-2) &&  \
        isValid){
      computed = true;
      int left = prev[W];
      int up = prev[tx];
      int right = prev[E];
      int shortest = MIN(left, up);
      shortest = MIN(shortest, right);
      int index = cols*(startStep+i)+xidx;
      result[tx] = shortest + gpuWall[index];

    }
    __syncthreads();
    if(i==iteration-1)
      break;
    if(computed)	 //Assign the computation range
      prev[tx]= result[tx];
    __syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
  }

  // update the global memory
  // after the last iteration, only threads coordinated within the 
  // small block perform the calculation and switch on ``computed''
  if (computed){
    gpuResults[xidx]=result[tx];		
  }
}

int main(int argc, char **argv) {
  hipSetDevice(1);
  srand(2013);
  unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;

  uiWA = WA * iSizeMultiple;
  uiHA = HA * iSizeMultiple;
  uiWB = WB * iSizeMultiple;
  uiHB = HB * iSizeMultiple;
  uiWC = WC * iSizeMultiple;
  uiHC = HC * iSizeMultiple;

  // allocate host memory for matrices A and B
  unsigned int size_A = uiWA * uiHA;
  unsigned int mem_size_A = sizeof(float) * size_A;
  float* h_A = (float*)malloc(mem_size_A);
  unsigned int size_B = uiWB * uiHB;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float* h_B = (float*)malloc(mem_size_B);

  // initialize host memory
  randomInit(h_A, size_A);
  randomInit(h_B, size_B);

  // allocate device memory
  float* d_A, *d_B, *d_C;
  unsigned int size_C = uiWC * uiHC;
  unsigned int mem_size_C = sizeof(float) * size_C;

  // allocate host memory for the result
  float* h_C      = (float*) malloc(mem_size_C);
  float* h_CUBLAS = (float*) malloc(mem_size_C);

  checkCudaErrors(hipMalloc((void**) &d_A, mem_size_A));
  checkCudaErrors(hipMalloc((void**) &d_B, mem_size_B));

  // copy host memory to device
  checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) );
  checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) );

  checkCudaErrors(hipMalloc((void**) &d_C, mem_size_C));

  pyramid_height=1;
  pf_cols=pf_NBLOCKS*pf_BLOCK_SIZE;
  pf_rows=2;//iteration=rows-1
  init(argc, argv);
  /* --------------- pyramid parameters --------------- */
  int borderCols = (pyramid_height)*HALO;
  int smallBlockCol = pf_BLOCK_SIZE-(pyramid_height)*HALO*2;
  printf("pf_cols=%d %d smallBlockcol\n",pf_cols,pf_rows,smallBlockCol);
  int blockCols = pf_cols/smallBlockCol+((pf_cols%smallBlockCol==0)?0:1);

  printf("pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",\
      pyramid_height, pf_cols, borderCols, pf_BLOCK_SIZE, blockCols, smallBlockCol);

  int *gpuWall, *gpuResult[2];
  int pf_size = pf_rows*pf_cols;

  hipMalloc((void**)&gpuResult[0], sizeof(int)*pf_cols);
  hipMalloc((void**)&gpuResult[1], sizeof(int)*pf_cols);
  hipMemcpy(gpuResult[0], pf_data, sizeof(int)*pf_cols, hipMemcpyHostToDevice);
  hipMalloc((void**)&gpuWall, sizeof(int)*(pf_size-pf_cols));
  hipMemcpy(gpuWall, pf_data+pf_cols, sizeof(int)*(pf_size-pf_cols), hipMemcpyHostToDevice);

  int t = 0;

  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float kernel_time = 0.0f;

  hipEventRecord(kernel_start, 0);

  hipStream_t stream[2];
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);

  int iters[2] = {1, 1};
  int finished[2] = {0, 0};
  // setup execution parameters
  dim3 mm_grid(mm_GRID_X, mm_GRID_Y);
  dim3 mm_block(mm_BLOCK_SIZE, mm_BLOCK_SIZE);

  mm_kernel<<< mm_grid, mm_block, 0, stream[mm_STREAM]>>>(d_C, d_A, d_B, uiWA, uiWB);

  int pf_block=(pf_BLOCK_SIZE);
  int pf_grid=(blockCols);  

  pf_kernel<<<pf_grid, pf_block, 0, stream[pf_STREAM]>>>(
      MIN(pyramid_height, pf_rows-t-1), 
      gpuWall, gpuResult[0], gpuResult[1],
      pf_cols,pf_rows, t, borderCols);


hipStreamSynchronize(stream[mm_STREAM]);
hipStreamSynchronize(stream[pf_STREAM]);

  for (int i = 0; i < 2; ++i)
    hipStreamDestroy(stream[i]);

  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);

  // get elapsed time
  kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
  kernel_time *= 1.e-3; // Convert to seconds

  cout << "kernel exe time: " << kernel_time << endl;
  // copy result from device to host
  checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost) );

  // compute reference solution
  float* reference = (float*)malloc(mem_size_C);
  computeGold(reference, h_A, h_B, uiHA, uiWA, uiWB);

  // check result (matrixMul)
  bool resCUDA = sdkCompareL2fe(reference, h_C, size_C, 1.0e-6f);
  printf("CUDA matrixMul compares %s\n\n", (true == resCUDA) ? "passed" : "FAIL");

//   ofstream f1("mm_correct.txt");
//   for(int i=0; i<size_C; ++i)
//     f1 << reference[i] << endl;
//   f1.close();
// 
//   ofstream f2("mm_gpu.txt");
//   for(int i=0; i<size_C; ++i)
//     f2 << h_C[i] << endl;
//   f2.close();


  // clean up memory
  free(h_A);
  free(h_B);
  free(h_C);
  free(reference);
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));


  hipMemcpy(pf_result, gpuResult[1], sizeof(int)*pf_cols, hipMemcpyDeviceToHost);

  check_kernel(MIN(pyramid_height, pf_rows-t-1), pf_data+pf_cols, pf_data, pf_result
      ,pf_cols,pf_rows, t, borderCols,pf_grid, pf_BLOCK_SIZE);

  hipFree(gpuWall);
  hipFree(gpuResult[0]);
  hipFree(gpuResult[1]);

  delete [] pf_data;
  delete [] wall;
  delete [] pf_result;

  return 0;
}

