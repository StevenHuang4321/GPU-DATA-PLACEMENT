#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
* This sample implements a separable convolution filter
* of a 2D image with an arbitrary kernel.
*/

// CUDA runtime
#include <hip/hip_runtime.h>

// Utilities and system includes
//#include <helper_functions.h>
#include <hip/hip_runtime.h>
#include "../include/common.h"
//#include <ctime.h>
#include <time.h>

#define KERNEL_RADIUS 8
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)
int line[1000000][6];
int yy=0;
__constant__ float c_Kernel[KERNEL_LENGTH];
void setConvolutionKernel(float *h_Kernel)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}


////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
#define   ROWS_BLOCKDIM_X 16
#define   ROWS_BLOCKDIM_Y 4
#define ROWS_RESULT_STEPS 8
#define   ROWS_HALO_STEPS 1
void convolutionRowsKernel_CPU(
    float *d_Dst,
    float *d_Src,
    float *kernel,
    int imageW,
    int imageH,
    int pitch
)
{FILE *f = fopen("hha.txt","w");
     float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    //Offset to the left halo edge
    for(int tx=0;tx<ROWS_BLOCKDIM_X;tx++)
     for(int ty = 0;ty<ROWS_BLOCKDIM_Y;ty++)
{


    const int baseX = (0 * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + tx;
    const int baseY = 0 * ROWS_BLOCKDIM_Y + ty;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Load main data
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        s_Data[ty][tx + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
    //fprintf(f,"1 0 0 %d %d %d\n",i-ROWS_HALO_STEPS,ty*ROWS_BLOCKDIM_X+tx,baseY*pitch+baseX+i*ROWS_BLOCKDIM_X);
    line[yy][0]=1;line[yy][1]=0;line[yy][2]=0;line[yy][3]=i-ROWS_HALO_STEPS;line[yy][4]=ty*ROWS_BLOCKDIM_X+tx;line[yy][5]=baseY*pitch+baseX+i*ROWS_BLOCKDIM_X;  
  yy++;
  }

    //Load left halo
#pragma unroll

    for (int i = 0; i < ROWS_HALO_STEPS; i++)
    {
        s_Data[ty][tx + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
 // fprintf(f,"1 0 1 %d %d %d\n",i,ty*ROWS_BLOCKDIM_X+tx,baseY*pitch+baseX+i*ROWS_BLOCKDIM_X);  
  line[yy][0]=1;line[yy][1]=0;line[yy][2]=1;line[yy][3]=i;line[yy][4]=ty*ROWS_BLOCKDIM_X+tx;line[yy][5]=baseY*pitch+baseX+i*ROWS_BLOCKDIM_X;
  yy++;

   }

    //Load right halo
#pragma unroll

    for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    {
        s_Data[ty][tx + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
  //fprintf(f,"1 0 2 %d %d %d\n",(i-ROWS_HALO_STEPS-ROWS_RESULT_STEPS),ty*ROWS_BLOCKDIM_X+tx,baseY*pitch+baseX+i*ROWS_BLOCKDIM_X);  
    line[yy][0]=1;line[yy][1]=0;line[yy][2]=2;line[yy][3]=i-ROWS_HALO_STEPS;line[yy][4]=ty*ROWS_BLOCKDIM_X+tx;line[yy][5]=baseY*pitch+baseX+i*ROWS_BLOCKDIM_X;
  yy++;
 
 }
 
}
    //Compute and store results
 //   __syncthreads();
#pragma unroll
  for(int tx=0;tx<ROWS_BLOCKDIM_X;tx++)
     for(int ty = 0;ty<ROWS_BLOCKDIM_Y;ty++)
{
    const int baseX = (0 * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + tx;
    const int baseY = 0 * ROWS_BLOCKDIM_Y + ty;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        float sum = 0;

#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += kernel[KERNEL_RADIUS - j] * s_Data[ty][tx + i * ROWS_BLOCKDIM_X + j];
   // fprintf(f,"0 0 0 %d %d %d\n",(i-ROWS_HALO_STEPS)*(2*KERNEL_RADIUS+1)+j+KERNEL_RADIUS,ty*ROWS_BLOCKDIM_X+tx,KERNEL_RADIUS - j);    
    line[yy][0]=0;line[yy][1]=0;line[yy][2]=0;line[yy][3]=(i-ROWS_HALO_STEPS)*(2*KERNEL_RADIUS+1)+j+KERNEL_RADIUS;line[yy][4]=ty*ROWS_BLOCKDIM_X+tx;line[yy][5]=baseY*pitch+baseX+i*ROWS_BLOCKDIM_X;
  yy++;
 
   }

        d_Dst[i * ROWS_BLOCKDIM_X] = sum;
  fprintf(f,"2 1 0 %d %d %d\n",(i-ROWS_HALO_STEPS),ty*ROWS_BLOCKDIM_X+tx,baseY*pitch+baseX+i*ROWS_BLOCKDIM_X);  
  
line[yy][0]=2;line[yy][1]=1;line[yy][2]=0;line[yy][3]=(i-ROWS_HALO_STEPS);line[yy][4]=ty*ROWS_BLOCKDIM_X+tx;line[yy][5]=baseY*pitch+baseX+i*ROWS_BLOCKDIM_X;
yy++;
}
}
}
__global__ void convolutionRowsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    //Offset to the left halo edge
    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Load main data
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
    }

    //Load left halo
#pragma unroll

    for (int i = 0; i < ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Load right halo
#pragma unroll

    for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        float sum = 0;

#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
        }

        d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}

void convolutionRowsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH
)
{
    assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
    assert(imageH % ROWS_BLOCKDIM_Y == 0);

    dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), imageH / ROWS_BLOCKDIM_Y);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

    convolutionRowsKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageW
    );
    getLastCudaError("convolutionRowsKernel() execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
#define   COLUMNS_BLOCKDIM_X 16
#define   COLUMNS_BLOCKDIM_Y 8
#define COLUMNS_RESULT_STEPS 8
#define   COLUMNS_HALO_STEPS 1

__global__ void convolutionColumnsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{
    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Main data
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * pitch];
    }

    //Upper halo
#pragma unroll

    for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Lower halo
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        float sum = 0;
#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
        }

        d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
    }
}

void convolutionColumnsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH
)
{
    assert(COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % COLUMNS_BLOCKDIM_X == 0);
    assert(imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0);

    dim3 blocks(imageW / COLUMNS_BLOCKDIM_X, imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);

    convolutionColumnsKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageW
    );
    getLastCudaError("convolutionColumnsKernel() execution failed\n");
}

void convolutionRowCPU(
    float *h_Dst,
    float *h_Src,
    float *h_Kernel,
    int imageW,
    int imageH,
    int kernelR
)
{
    for (int y = 0; y < imageH; y++)
        for (int x = 0; x < imageW; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = x + k;

                if (d >= 0 && d < imageW)
                    sum += h_Src[y * imageW + d] * h_Kernel[kernelR - k];
            }

            h_Dst[y * imageW + x] = sum;
        }
}



////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(
    float *h_Dst,
    float *h_Src,
    float *h_Kernel,
    int imageW,
    int imageH,
    int kernelR
)
{
    for (int y = 0; y < imageH; y++)
        for (int x = 0; x < imageW; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = y + k;

                if (d >= 0 && d < imageH)
                    sum += h_Src[d * imageW + x] * h_Kernel[kernelR - k];
            }

            h_Dst[y * imageW + x] = sum;
        }
}



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // start logs
    printf("[%s] - Starting...\n", argv[0]);

    float
    *h_Kernel,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;

    float
    *d_Input,
    *d_Output,
    *d_Buffer;


    const int imageW = 3072;
    const int imageH = 3072;
    const int iterations = 16;

    struct timespec t1,t2,t3,t4,t5,t6;
    clock_gettime(CLOCK_MONOTONIC,&t5);

    //Use command-line specified CUDA device, otherwise use device with highest Gflops/s
    //findCudaDevice(argc, (const char **)argv);

    

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    h_Kernel    = (float *)malloc(KERNEL_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));
    srand(200);

    for (unsigned int i = 0; i < KERNEL_LENGTH; i++)
    {
        h_Kernel[i] = (float)(rand() % 16);
    }

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
        h_Input[i] = (float)(rand() % 16);
    }

    printf("Allocating and initializing CUDA arrays...\n");
    checkCudaErrors(hipMalloc((void **)&d_Input,   imageW * imageH * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Output,  imageW * imageH * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Buffer , imageW * imageH * sizeof(float)));

    setConvolutionKernel(h_Kernel);
    checkCudaErrors(hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice));

    printf("Running GPU convolution (%u identical iterations)...\n\n", iterations);

    for (int i = -1; i < iterations; i++)
    {
        //i == -1 -- warmup iteration
        if (i == 0)
        {
            checkCudaErrors(hipDeviceSynchronize());
           // clock_gettime(CLOCK_MONOTONIC,&t1);
        }
if(i==1) clock_gettime(CLOCK_MONOTONIC,&t1);
        convolutionRowsGPU(
            d_Buffer,
            d_Input,
            imageW,
            imageH
        );
hipDeviceSynchronize();
if(i==1) clock_gettime(CLOCK_MONOTONIC,&t2);
if(i==1)  clock_gettime(CLOCK_MONOTONIC,&t3);
if(i==1)convolutionRowsKernel_CPU(
            h_Buffer,
            h_Input,
           h_Kernel,  
            imageW,
            imageH,imageW
        );
if(i==1)  clock_gettime(CLOCK_MONOTONIC,&t4);

        convolutionColumnsGPU(
            d_Output,
            d_Buffer,
            imageW,
            imageH
        );
    }

    checkCudaErrors(hipDeviceSynchronize());
    //clock_gettime(CLOCK_MONOTONIC,&t2);
    double gpuTime = ((t2.tv_sec-t1.tv_sec)+ (t2.tv_nsec-t1.tv_nsec)/1.e9);/// (double)iterations;
    printf("convolutionSeparable, Throughput = %.4f MPixels/sec, Time = %.5f s, Size = %u Pixels, NumDevsUsed = %i, Workgroup = %u\n",
           (1.0e-6 * (double)(imageW * imageH)/ gpuTime), gpuTime, (imageW * imageH), 1, 0);
    double cpuTime = ((t4.tv_sec-t3.tv_sec)+ (t4.tv_nsec-t3.tv_nsec)/1.e9);
    printf("CPUTIME: %f\n",cpuTime);
    printf("\nReading back GPU results...\n\n");
    checkCudaErrors(hipMemcpy(h_OutputGPU, d_Output, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost));

    printf("Checking the results...\n");
    printf(" ...running convolutionRowCPU()\n");
    convolutionRowCPU(
        h_Buffer,
        h_Input,
        h_Kernel,
        imageW,
        imageH,
        KERNEL_RADIUS
    );

    printf(" ...running convolutionColumnCPU()\n");
    convolutionColumnCPU(
        h_OutputCPU,
        h_Buffer,
        h_Kernel,
        imageW,
        imageH,
        KERNEL_RADIUS
    );

    printf(" ...comparing the results\n");
    double sum = 0, delta = 0;

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
        delta += (h_OutputGPU[i] - h_OutputCPU[i]) * (h_OutputGPU[i] - h_OutputCPU[i]);
        sum   += h_OutputCPU[i] * h_OutputCPU[i];
    }

    double L2norm = sqrt(delta / sum);
    printf(" ...Relative L2 norm: %E\n\n", L2norm);
    printf("Shutting down...\n");


    checkCudaErrors(hipFree(d_Buffer));
    checkCudaErrors(hipFree(d_Output));
    checkCudaErrors(hipFree(d_Input));
    free(h_OutputGPU);
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Kernel);

    

    hipDeviceReset();
clock_gettime(CLOCK_MONOTONIC,&t6);
printf("Total Time : %f\n",(t6.tv_sec-t5.tv_sec)+ (t6.tv_nsec-t5.tv_nsec)/1.e9);

    if (L2norm > 1e-6)
    {
        printf("Test failed!\n");
        exit(EXIT_FAILURE);
    }

    printf("Test passed\n");
    exit(EXIT_SUCCESS);
}
