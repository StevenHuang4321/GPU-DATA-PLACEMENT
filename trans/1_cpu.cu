#include "hip/hip_runtime.h"

#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <list>
#include <map>
#include <math.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <algorithm>
#include <iterator>
#include <fstream>
#include "../include/common.h"
#define K 1
using namespace std;


#define trans_BLOCK_SIZE 16

#define trans_GRID_X 1600
#define trans_GRID_Y 480
#define trans_NBLOCKS (trans_GRID_X*trans_GRID_Y) 

texture<float,2,hipReadModeElementType> tex_A;
void computeGold( float* reference, float* idata,
            const unsigned int size_x, const unsigned int size_y )
{
  // transpose matrix
  for( unsigned int y = 0; y < size_y; ++y)
  {
    for( unsigned int x = 0; x < size_x; ++x)
    {
      reference[(x * size_y) + y] = idata[(y * size_x) + x];
    }
  }
}

void trans_checkResults(float *h_idata, float *h_odata, int width, int height)
{
 /* // compute reference solution
  int trans_mem_size = width * height *sizeof(float);
  float* reference = (float*) malloc( trans_mem_size);

  computeGold( reference, h_idata, width, height);

  for(int i=0; i<width*height; ++i)
  {
    if(reference[i] != d_odata[i]) {
      fprintf(stderr, "Failed! i = %d\n", i);
      return;
    }
  }
  printf("Test passed!\n");*/
FILE *f = fopen("hha.txt","w");
#pragma omp for  
for(unsigned int xIndex =0;xIndex<16;xIndex++){
   for (unsigned int yIndex =0;yIndex<16;yIndex++){
    if (xIndex < width && yIndex < height)
    {
       unsigned int index_in  = xIndex + width * yIndex;
       unsigned int index_out = yIndex + height * xIndex;
      h_odata[index_out]=h_idata[index_in];
  //    if( h_odata[index_out] != h_idata[index_in]) {printf("failed!%d %d %f %f \n ",xIndex,yIndex,h_odata[index_out],h_idata[index_in]);return;}
    fprintf(f,"1 0 0 0 %d %d\n",xIndex+16*yIndex,index_in);   
 }
    }
    }
    printf("GOOD!trans passed\n");
    return;
}

__global__ void trans_kernel(float *odata, float* idata, int width, int height)
{
   unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

   if (xIndex < width && yIndex < height)
   {
       unsigned int index_in  = xIndex + width * yIndex;
       unsigned int index_out = yIndex + height * xIndex;
       odata[index_out] = tex2D(tex_A,index_in%width,index_in/width);//idata[index_in];
   }
}

int main(int argc, char **argv) {
  hipSetDevice(2);
  srand(2013);
struct timespec start,end;
clock_gettime(CLOCK_MONOTONIC,&start);
  const unsigned int trans_size_x = trans_GRID_X * trans_BLOCK_SIZE;
  const unsigned int trans_size_y = trans_GRID_Y * trans_BLOCK_SIZE;

  // size of memory required to store the matrix
  const unsigned int trans_mem_size = sizeof(float) * trans_size_x * trans_size_y;

  // allocate host memory
  float* h_trans_idata = (float*) malloc(trans_mem_size);
  // initalize the memory
  for( unsigned int i = 0; i < (trans_size_x * trans_size_y); ++i)
  {
      h_trans_idata[i] = (float) i;    // rand();
  }

  // allocate device memory
  float* d_trans_idata;
  float* d_trans_odata;
  hipMalloc( (void**) &d_trans_idata, trans_mem_size);
  hipMalloc( (void**) &d_trans_odata, trans_mem_size);

  // copy host memory to device
  hipMemcpy( d_trans_idata, h_trans_idata, trans_mem_size, hipMemcpyHostToDevice);

 hipChannelFormatDesc channelDescA =  hipCreateChannelDesc<float>();
  hipArray* A_Array;
 hipMallocArray(&A_Array, &channelDescA, trans_size_x,trans_size_y);
 hipMemcpyToArray(A_Array, 0, 0, h_trans_idata, trans_mem_size,
                      hipMemcpyHostToDevice);
 tex_A.addressMode[0] = hipAddressModeWrap;
    tex_A.addressMode[1] = hipAddressModeWrap;
    tex_A.filterMode     = hipFilterModePoint;
 hipBindTextureToArray(tex_A, A_Array, channelDescA);

  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float kernel_time = 0.0f;

  hipEventRecord(kernel_start, 0);
  // setup execution parameters
  dim3 trans_grid(trans_GRID_X, trans_GRID_Y, 1);
  dim3 trans_block(trans_BLOCK_SIZE, trans_BLOCK_SIZE, 1);
 

  trans_kernel<<<trans_grid, trans_block>>>(d_trans_odata, d_trans_idata, trans_size_x, trans_size_y);

  hipDeviceSynchronize();

  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);

  // get elapsed time
  kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
  kernel_time *= 1.e-3; // Convert to seconds
  
  cout << "kernel exe time: " << kernel_time << endl;
  float* h_trans_odata = (float*) malloc(trans_mem_size);
  hipMemcpy( h_trans_odata, d_trans_odata, trans_mem_size, hipMemcpyDeviceToHost);


  // check result
  struct timespec t1, t2;
  clock_gettime(CLOCK_MONOTONIC,&t1);
  trans_checkResults(h_trans_idata, h_trans_odata, trans_size_x, trans_size_y);
  clock_gettime(CLOCK_MONOTONIC,&t2);
double kernel_timehh = (t2.tv_sec-t1.tv_sec)*1.e9+t2.tv_nsec-t1.tv_nsec;
  printf("Kernel time %f\n",kernel_timehh*1.e-9);
clock_gettime(CLOCK_MONOTONIC,&end);
double total_time = (end.tv_sec-start.tv_sec)*1.e9+end.tv_nsec-start.tv_nsec;
  printf("Kernel time %f\n",total_time*1.e-9);
  return 0;
}

