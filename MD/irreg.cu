#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <iostream>
//#define n 256000

using namespace std;
__constant__ int d_B[64000];
int CPU_results(int *C, int *B,int *A,int N)
{
 for(int i=0;i<N;i++)
  C[B[i]]=A[i];
 return 0;
}
int check_results(int *C, int *B, int *A, int N)
{
 for(int i=0; i<N; i++)
 { if(C[B[i]]!=A[i]) 
  {
   cout<<i<<endl;
   cout<<A[i]<<" "<<C[B[i]]<<endl;
   printf("Sorry! Checking Failed!\n");
   return 0;
  }
 }
 printf("Good! Checking Passed!\n");
 return 1;
}

__global__ void kernel(int *d_C, int *d_A, int N)
{
 int tid = blockIdx.x * blockDim.x + threadIdx.x;
 if(tid >= N) return;
 int x = d_B[d_B[tid]];
}

int main(int argc, char *argv[])
{
 int N=atoi(argv[1]);
 int *A, *B, *C, *d_A, *d_B, *d_C;
 A=(int *)malloc(N*sizeof(int));
 B=(int *)malloc(N*sizeof(int));
 C=(int *)malloc(N*sizeof(int));
 hipMalloc((void **)&d_A, N*sizeof(int));
 hipMalloc((void **)&d_B, N*sizeof(int));
 hipMalloc((void **)&d_C, N*sizeof(int));
 srand(2013);
 vector<int> BV(N);
 for(int i=0; i<N; i++)
 {
  A[i]=rand()%N;
  //cout<<"A["<<i<<"]="<<A[i]<<endl;
  BV[i]=i;//rand()%N;
  }
  random_shuffle(BV.begin(),BV.end());
 for(int i=0;i<N;i++)
 B[i]=BV[i];
 
 hipMemcpy(d_A,A,N*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpyToSymbol(HIP_SYMBOL(d_B),B,N*sizeof(int));
 int blocks= 256;
 struct timespec time_start, time_end;
 clock_gettime(CLOCK_MONOTONIC,&time_start);
 kernel<<<(N+255)/256,blocks>>>(d_C,d_A,N);
 hipDeviceSynchronize();
 clock_gettime(CLOCK_MONOTONIC,&time_end);
 double kernel_time=(time_end.tv_sec-time_start.tv_sec)*1.e9+time_end.tv_nsec-time_start.tv_nsec;
 cout<<"GPU kernel time= "<<kernel_time*1.e-9<<endl;
 //for(int i=0;i<N;i++)
 //cout<<"C "<<i<<"="<<C[i]<<endl;
 clock_gettime(CLOCK_MONOTONIC,&time_start);
 //CPU_results(C,B,A,N);
 clock_gettime(CLOCK_MONOTONIC,&time_end);
 kernel_time=(time_end.tv_sec-time_start.tv_sec)*1.e9+time_end.tv_nsec-time_start.tv_nsec;
 cout<<"CPU time= "<<kernel_time*1.e-9<<endl;
 hipMemcpy(C,d_C,N*sizeof(int),hipMemcpyDeviceToHost);
 //check_results(C,B,A,N);
 return 0;
}
