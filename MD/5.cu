#include "hip/hip_runtime.h"

#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <list>
#include <map>
#include <math.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <algorithm>
#include <iterator>
#include <fstream>
#include "../include/common.h"
#define K 1
using namespace std;

#define md_NBLOCKS 16*6*2*2
#define md_BLOCK_SIZE 256
//#define md_SUPER_BLOCKS_PER_SM 4
//int md_SUPER_BLOCKS_PER_SM = 4;
texture<int,1,hipReadModeElementType> tex_neibor;
const int md_nBlksPerCluster = 16; 
const int md_nAtom = md_BLOCK_SIZE * md_NBLOCKS;
const float md_cutsq = 16.0f;
const int md_maxNeighbors = 30;
const double md_domainEdge   = 20.0; // Edge length of the cubic domain
const float md_lj1 = 1.5;
const float md_lj2 = 2.0;
const float  md_EPSILON      = 0.1f; // Relative Error between CPU/GPU


inline int * md_myBuildNeighborList_blkSchedule(const int nAtom, float3* position,
        int* neighborList, int blockSz)
{
  vector<int> atomInds(nAtom);
  vector<int> blkInds((nAtom+blockSz-1)/blockSz);
  for(int i=0; i<blkInds.size(); ++i)
    blkInds[i] = i;
  random_shuffle(blkInds.begin(), blkInds.end());

  int *blkOrder = (int*)malloc(blkInds.size()*sizeof(int));
  for(int i=0; i<blkInds.size(); ++i)
    blkOrder[i] = blkInds[i];

  int j=0; 
  for(vector<int>::iterator it=blkInds.begin(); it!=blkInds.end(); ++it)
  {
    int blkInd = *it;
    for(int i=0; i<blockSz; ++i)
      atomInds[j++] = blkInd*blockSz + i;
  }
  int superBlockSz = blockSz * md_nBlksPerCluster;
  // Build Neighbor List
  for (int i = 0; i < nAtom; i++)
  {
    int start = i - i%superBlockSz; //difference is here
    int end = i + (superBlockSz - i%superBlockSz)-1;

    int nNeighbors = 0;
    do {
      int j = start + rand() % superBlockSz;
      if (i == j || j>=nAtom) continue; // An atom cannot be its own neighbor
      neighborList[nNeighbors*nAtom + atomInds[i]] = atomInds[j];
      nNeighbors ++; 
    } while(nNeighbors<md_maxNeighbors);

  }
  return blkOrder;
}

bool md_checkResults(float3* d_force, float3* position, int *neighList, int nAtom)
{
  for (int i = 0; i < nAtom; i++)
  {
    float3 ipos = position[i];
    float3 f = {0.0f, 0.0f, 0.0f};
    int j = 0;
    while (j < md_maxNeighbors)
    {
      int jidx = neighList[j*nAtom + i];
      float3 jpos = position[jidx];
      //if(i == 196)
      // printf("jidx = %d, ipos.x = %f, jpos.x = %f\n", jidx, ipos.x, jpos.x);
      // Calculate distance
      float delx = ipos.x - jpos.x;
      float dely = ipos.y - jpos.y;
      float delz = ipos.z - jpos.z;
      float r2inv = delx*delx + dely*dely + delz*delz;

      // If distance is less than cutoff, calculate force
      if (r2inv < md_cutsq) {

        r2inv = 1.0f/r2inv;
        float r6inv = r2inv * r2inv * r2inv;
        float force = r2inv*r6inv*(md_lj1*r6inv - md_lj2);

        f.x += delx * force;
        f.y += dely * force;
        f.z += delz * force;
      }
      j++;
    }
    //if(i==0)
    //cerr << d_force[i].x << endl;
    // Check the results
    float diffx = (d_force[i].x - f.x) / d_force[i].x;
    float diffy = (d_force[i].y - f.y) / d_force[i].y;
    float diffz = (d_force[i].z - f.z) / d_force[i].z;
    float err = sqrt(diffx*diffx) + sqrt(diffy*diffy) + sqrt(diffz*diffz);
    if (err > (3.0 * md_EPSILON))
    {
      cout << "Test Failed, idx: " << i << " diff: " << err << "\n";
      cout << "f.x: " << f.x << " df.x: " << d_force[i].x << "\n";
      cout << "f.y: " << f.y << " df.y: " << d_force[i].y << "\n";
      cout << "f.z: " << f.z << " df.z: " << d_force[i].z << "\n";
      cout << "Test FAILED\n";
      return false;
    }
  }
  cout << "Test Passed\n";
  return true;
}
__global__ void md_kernel(float3*  force3,
                                 const float3* __restrict__ position,
                                 const int neighCount,
                                 int*  neighList,
                                 const float cutsq,
                                 const float lj1,
                                 const float lj2,
                                 const int inum)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  // Position of this thread's atom
  float3 ipos = position[idx];

  // Force accumulator
  float3 f = {0.0f, 0.0f, 0.0f};


  int j = 0; 
  while (j < neighCount)
  {
    int jidx = tex1Dfetch(tex_neibor,j*inum + idx);
    float3 jpos;
    jpos = position[jidx];

    // Calculate distance
    float delx = ipos.x - jpos.x;
    float dely = ipos.y - jpos.y;
    float delz = ipos.z - jpos.z;
    float r2inv = delx*delx + dely*dely + delz*delz;

    // If distance is less than cutoff, calculate force
    // and add to accumulator
    if (r2inv < cutsq)
    {    
      r2inv = 1.0f/r2inv;
      float r6inv = r2inv * r2inv * r2inv;
      float force = r2inv*r6inv*(lj1*r6inv - lj2);

      f.x += delx * force;
      f.y += dely * force;
      f.z += delz * force;
    }    
    j++; 
  }

  // store the results
  force3[idx] = f;
//if (threadIdx.x==0) atomicAdd(d_flag,1);
}

int main(int argc, char **argv) {
//  hipSetDevice(1);
  srand(2013);
  float3* md_position;
  float3* md_force;
  int* md_neighborList;

  hipHostMalloc((void**)&md_position, md_nAtom*sizeof(float3));
  hipHostMalloc((void**)&md_force,    md_nAtom*sizeof(float3));
  hipHostMalloc((void**)&md_neighborList, md_nAtom*md_maxNeighbors*sizeof(int));

  // Allocate device memory for position and force
  float3* d_md_force;
  float3* d_md_position;
  hipMalloc((void**)&d_md_force, md_nAtom*sizeof(float3));
  hipMemset(d_md_force, 0, md_nAtom*sizeof(float3));
  hipMalloc((void**)&d_md_position, md_nAtom*sizeof(float3));

  // Allocate device memory for neighbor list
  int* d_md_neighborList;
  hipMalloc((void**)&d_md_neighborList, md_nAtom*md_maxNeighbors*sizeof(int));

  for (int i = 0; i < md_nAtom; i++)
  {
    md_position[i].x = (float)(drand48() * md_domainEdge);
    md_position[i].y = (float)(drand48() * md_domainEdge);
    md_position[i].z = (float)(drand48() * md_domainEdge);
  }

  md_myBuildNeighborList_blkSchedule(md_nAtom, md_position,
          md_neighborList, md_BLOCK_SIZE);

  hipMemcpy(d_md_neighborList, md_neighborList, md_maxNeighbors*md_nAtom*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_md_position, md_position, md_nAtom*sizeof(float3), hipMemcpyHostToDevice);

hipBindTexture(0,tex_neibor,d_md_neighborList,md_maxNeighbors*md_nAtom* sizeof(int));
  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float kernel_time = 0.0f;

  hipEventRecord(kernel_start, 0);

  int md_gridSize  = (md_nAtom-1+md_BLOCK_SIZE) / md_BLOCK_SIZE;
    md_kernel<<<md_gridSize, md_BLOCK_SIZE>>>
                  (d_md_force, d_md_position, md_maxNeighbors, d_md_neighborList,
                   md_cutsq, md_lj1, md_lj2, md_nAtom);


  hipDeviceSynchronize();

  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);

  // get elapsed time
  kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
  kernel_time *= 1.e-3; // Convert to seconds
  
  cout << "kernel exe time: " << kernel_time << endl;

  hipMemcpy(md_force, d_md_force, md_nAtom*sizeof(float3),
            hipMemcpyDeviceToHost);

  md_checkResults(md_force, md_position, md_neighborList, md_nAtom);


  return 0;
}

